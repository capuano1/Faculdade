#include "hip/hip_runtime.h"
/*
ABANDONADO E REESCRITO -> NOVO ARQUIVO: FinalCuda2.cu
Salvo por motivos de segurança, caso o outro jeito desse errado
*/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define N 2000  // Tamanho da grade
#define B 256 //Quantos blocos
#define T 2000 // Número de iterações
#define D 0.1  // Coeficiente de difusão
#define DELTA_T 0.01
#define DELTA_X 1.0

__global__ void diff_eq(double* C, double* C_new) {
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            C_new[j + (i*N)] = C[j + (i*N)] + D * DELTA_T * (
                (C[j+1 + (i*N)] + C[j-1 + (i*N)] + C[j + ((i+1)*N)] + C[j + ((i-1)*N)] - 4 * C[j + (i*N)]) / (DELTA_X * DELTA_X)
            );
        }
    }
    __syncthreads();
    //Separar em dois
    double difmedio = 0.;
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            difmedio += fabs(C_new[j + (i*N)] - C[j + (i*N)]);
            C[j + (i*N)] = C_new[j + (i*N)];
        }
    }
    //if ((t%100) == 0) printf("interacao %d - diferenca=%g\n", t, difmedio/((N-2)*(N-2)));
    __syncthreads();
}

//Melhor a diff_eq ser a funcao que vai, no host, preparar tudo para o device e chamar as duas funcoes do device

int main() {
    double *C = (double*)malloc(N * N * sizeof(double));      // Concentração inicial
    double *C_new = (double*)malloc(N * N * sizeof(double));  // Concentração para a próxima iteração
    double *C_d;
    double *C_new_d;

    if (C == NULL || C_new == NULL) {
        fprintf(stderr, "Erro ao alocar memória para as linhas\n");
        return 1;
    }

    memset(C, 0, N * N * sizeof(double));
    memset(C_new, 0, N * N * sizeof(double));

    hipMalloc((void**)&C_d, N*N*sizeof(double));
    hipMalloc((void**)&C_new_d, N*N*sizeof(double));

    // Inicializar uma concentração alta no centro
    C[(N/2) + (N*(N/2))] = 1.0;

    hipMemcpy(C_d, C, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(C_new_d, C_new, N * N * sizeof(double), hipMemcpyHostToDevice);

    clock_t tempo;
    tempo = clock();

    // Executar a equação de difusão
    // for mais externo foi puxado para cá pois ele não é paralelizavel
    for (int i = 0; i < T; i++) {
        diff_eq(C, C_new);
    }

    // Exibir resultado para verificação
    printf("Concentracao final no centro: %f\n", C[(N/2) + (N*(N/2))]);

    // Liberar a memória alocada
    free(C);
    free(C_new);
    hipFree(C_d);
    hipFree(C_new_d);

    printf("Tempo: %f segundos\n", ((double)clock() - tempo)/CLOCKS_PER_SEC);

    return 0;
}
