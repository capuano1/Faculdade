
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define N 2000 // Tamanho da grade
#define T 2000 // Número de iterações
#define D 0.1 // Coeficiente de difusão
#define DELTA_T 0.01
#define DELTA_X 1.0
int BLOCK_SIZE = 0; // Threads per block -> no dim3 ele acaba fazendo dim3(x, y) = x*y threads, no caso, BLOCK_SIZE^2 threads.
// Ou seja, BLOCK_SIZE = sqrt(threads), vulgo, raiz quadrada do numero de threads
// Portanto, numero maximo seria 32, que vai dar 1024 threads

__global__ void calculateC(double* C, double* C_new) { 
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < N - 1 && j < N - 1) {
        C_new[j + i * N] = C[j + i * N] + D * DELTA_T * (
            (C[j + 1 + i * N] + C[j - 1 + i * N] + C[j + (i + 1) * N] + C[j + (i - 1) * N] - 4 * C[j + i * N]) / (DELTA_X * DELTA_X) );
    }
}

__global__ void updateC(double* C, double* C_new, double* difmedio) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < N - 1 && j < N - 1) {
        *difmedio += fabs(C_new[j + i * N] - C[j + i * N]);
        C[j + i * N] = C_new[j + i * N];
    }
}

void diff_eq(double* C, double* C_new) {
    double* d_C;
    double* d_C_new;
    double* d_difmedio;
    double difmedio;
    
    hipMalloc(&d_C, N * N * sizeof(double));
    hipMalloc(&d_C_new, N * N * sizeof(double));
    hipMalloc(&d_difmedio, sizeof(double));

    hipMemcpy(d_C, C, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C_new, C_new, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    for (int t = 0; t < T; t++) {
        hipMemset(d_difmedio, 0, sizeof(double));
        calculateC<<<gridSize, blockSize>>>(d_C, d_C_new);
        hipDeviceSynchronize();
        updateC<<<gridSize, blockSize>>>(d_C, d_C_new, d_difmedio);
        hipDeviceSynchronize();
        hipMemcpy(&difmedio, d_difmedio, sizeof(double), hipMemcpyDeviceToHost);
        /*if ((t % 100) == 0) {
            printf("interacao %d - diferenca=%g\n", t, difmedio / ((N - 2) * (N - 2)));
        }*/
    }
    
    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C_new, d_C_new, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_C);
    hipFree(d_C_new);
    hipFree(d_difmedio);
}

int main() {
    double* C = (double*)malloc(N * N * sizeof(double)); // Concentração inicial
    double* C_new = (double*)malloc(N * N * sizeof(double)); // Concentração para a próxima iteração
    FILE *tempos;
    
    if (C == NULL || C_new == NULL) {
        fprintf(stderr, "Erro ao alocar memória\n");
        return 1;
    }
    

    clock_t tempo;
    double temposReg[10];
    tempos = fopen("FinalCUDA-Tempo.txt", "w");
    memset(C, 0, N * N * sizeof(double));
    memset(C_new, 0, N * N * sizeof(double));
    for (; BLOCK_SIZE <= 32; BLOCK_SIZE += 4) {
        if (BLOCK_SIZE == 0) BLOCK_SIZE++; //Para fazer com 1 thread apenas
        
        double mediaTempo = 0, desvioPadraoTempo = 0;
        for (int z = 0; z < 10; z++) {
            tempo = clock();
    
            // Inicializar uma concentração alta no centro
            C[N / 2 * N + N / 2] = 1.0;
    
            // Executar a equação de difusão
            diff_eq(C, C_new);

            temposReg[z] = ((double)clock() - tempo)/CLOCKS_PER_SEC;
            mediaTempo += temposReg[z];
            memset(C, 0, N * N * sizeof(double));
            memset(C_new, 0, N * N * sizeof(double));
        }
        mediaTempo /= 10;
        for (int i = 0; i < 10; i++) {
            desvioPadraoTempo += pow((temposReg[i] - mediaTempo), 2);
        }
        desvioPadraoTempo /= 10;
        desvioPadraoTempo = sqrt(desvioPadraoTempo);
        fprintf(tempos, "%f %f\n", mediaTempo, desvioPadraoTempo);
        if (BLOCK_SIZE == 1) BLOCK_SIZE--; //Para desfazer a iteracao de 1 thread, voltar para 0 e ir em incrementos de 4 ate 32
    }
    
    // Liberar a memória alocada
    free(C);
    free(C_new);
    printf("Tempo: %f segundos\n", ((double)clock() - tempo) / CLOCKS_PER_SEC);
    return 0;
}