
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define N 2000 // Tamanho da grade
int T = 2000; // Número de iterações
#define D 0.1 // Coeficiente de difusão
#define DELTA_T 0.01
#define DELTA_X 1.0
#define BLOCK_SIZE 16 // Threads per block -> no dim3 ele acaba fazendo dim3(x, y) = x*y threads, no caso, BLOCK_SIZE^2 threads.
// Ou seja, BLOCK_SIZE = sqrt(threads), vulgo, raiz quadrada do numero de threads
// Portanto, numero maximo seria 32, que vai dar 1024 threads

__global__ void calculateC(double* C, double* C_new) { 
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < N - 1 && j < N - 1) {
        C_new[j + i * N] = C[j + i * N] + D * DELTA_T * (
            (C[j + 1 + i * N] + C[j - 1 + i * N] + C[j + (i + 1) * N] + C[j + (i - 1) * N] - 4 * C[j + i * N]) / (DELTA_X * DELTA_X) );
    }
}

__global__ void updateC(double* C, double* C_new, double* difmedio) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (i < N - 1 && j < N - 1) {
        *difmedio += fabs(C_new[j + i * N] - C[j + i * N]);
        C[j + i * N] = C_new[j + i * N];
    }
}

void diff_eq(double* C, double* C_new) {
    double* d_C;
    double* d_C_new;
    double* d_difmedio;
    double difmedio;
    
    hipMalloc(&d_C, N * N * sizeof(double));
    hipMalloc(&d_C_new, N * N * sizeof(double));
    hipMalloc(&d_difmedio, sizeof(double));

    hipMemcpy(d_C, C, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C_new, C_new, N * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    for (int t = 0; t < T; t++) {
        hipMemset(d_difmedio, 0, sizeof(double));
        calculateC<<<gridSize, blockSize>>>(d_C, d_C_new);
        hipDeviceSynchronize();
        updateC<<<gridSize, blockSize>>>(d_C, d_C_new, d_difmedio);
        hipDeviceSynchronize();
        hipMemcpy(&difmedio, d_difmedio, sizeof(double), hipMemcpyDeviceToHost);
        /*if ((t % 100) == 0) {
            printf("interacao %d - diferenca=%g\n", t, difmedio / ((N - 2) * (N - 2)));
        }*/
    }
    
    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(C_new, d_C_new, N * N * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_C);
    hipFree(d_C_new);
    hipFree(d_difmedio);
}

int main() {
    double* C = (double*)malloc(N * N * sizeof(double)); // Concentração inicial
    double* C_new = (double*)malloc(N * N * sizeof(double)); // Concentração para a próxima iteração
    FILE *dados;
    
    if (C == NULL || C_new == NULL) {
        fprintf(stderr, "Erro ao alocar memória\n");
        return 1;
    }
    
    dados = fopen("FinalCUDA-DadosCompatibilidade.txt", "w");
    for (; T < 2100; T++) {
        memset(C, 0, N * N * sizeof(double));
        memset(C_new, 0, N * N * sizeof(double));
    
        // Inicializar uma concentração alta no centro
        C[N / 2 * N + N / 2] = 1.0;
    
        // Executar a equação de difusão
        diff_eq(C, C_new);
    
        // Exibir resultado para verificação
        fprintf(dados, "%f\n", C[N / 2 * N + N / 2]);
    }

    fclose(dados);

    // Liberar a memória alocada
    free(C);
    free(C_new);
    return 0;
}